/* Two-level binary reduction solution.
 */


#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

// Number of times to run the test (for better timings accuracy):
#define NTESTS 10

// Number of threads in one block (possible range is 32...1024):
#define BLOCK_SIZE 256

// Total number of threads (total number of elements to process in the kernel):
// For simplicity, use a power of two:
#define NMAX 131072

#define NBLOCKS NMAX / BLOCK_SIZE

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))

// Input array (global host memory):
float h_A[NMAX];
// Copy of h_A on device:
__device__ float d_A[NMAX];
__device__ float d_min1[NBLOCKS];

__device__ float d_min;

/* Subtract the `struct timeval' values X and Y,
   storing the result in RESULT.
   Return 1 if the difference is negative, otherwise 0.  */

// It messes up with y!

int timeval_subtract(double *result, struct timeval *x, struct timeval *y)
{
    struct timeval result0;

    /* Perform the carry for the later subtraction by updating y. */
    if (x->tv_usec < y->tv_usec)
    {
        int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
        y->tv_usec -= 1000000 * nsec;
        y->tv_sec += nsec;
    }
    if (x->tv_usec - y->tv_usec > 1000000)
    {
        int nsec = (y->tv_usec - x->tv_usec) / 1000000;
        y->tv_usec += 1000000 * nsec;
        y->tv_sec -= nsec;
    }

    /* Compute the time remaining to wait.
       tv_usec is certainly positive. */
    result0.tv_sec = x->tv_sec - y->tv_sec;
    result0.tv_usec = x->tv_usec - y->tv_usec;
    *result = ((double)result0.tv_usec) / 1e6 + (double)result0.tv_sec;

    /* Return 1 if result is negative. */
    return x->tv_sec < y->tv_sec;
}

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

// Kernel(s) should go here:

// First step in binary reduction:
__global__ void MyKernel1()
{
    __shared__ float min[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Not needed, because NMAX is a power of two:
    //  if (i >= NMAX)
    //    return;

    min[threadIdx.x] = d_A[i];

    // To make sure all threads in a block have the sum[] value:
    __syncthreads();

    int nTotalThreads = blockDim.x; // Total number of active threads;
    // only the first half of the threads will be active.

    while (nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1); // divide by two

        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            min[threadIdx.x] = MIN(min[threadIdx.x],min[thread2]); // Pairwise comparison
        }
        __syncthreads();
        nTotalThreads = halfPoint; // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0)
    {
        d_min1[blockIdx.x] = min[0];
    }

    return;
}

// Second step in binary reduction (one block):
__global__ void MyKernel2()
{
    __shared__ float min[NBLOCKS];

    // Copying from global to shared memory:
    min[threadIdx.x] = d_min1[threadIdx.x];

    // To make sure all threads in a block have the sum[] value:
    __syncthreads();

    int nTotalThreads = blockDim.x; // Total number of active threads;
    // only the first half of the threads will be active.

    while (nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1); // divide by two

        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            min[threadIdx.x] = MIN(min[threadIdx.x],min[thread2]); // Pairwise comparison
        }
        __syncthreads();
        nTotalThreads = halfPoint; // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0)
    {
        d_min = min[0];
    }

    return;
}

int main(int argc, char **argv)
{
    struct timeval tdr0, tdr1, tdr;
    double restime, min0;
    float min;
    int devid, devcount, error;

    /* find number of device in current "context" */
    hipGetDevice(&devid);
    /* find how many devices are available */
    if (hipGetDeviceCount(&devcount) || devcount == 0)
    {
        printf("No CUDA devices!\n");
        exit(1);
    }
    else
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, devid);
        printf("Device count, devid: %d %d\n", devcount, devid);
        printf("Device: %s\n", deviceProp.name);
        printf("[deviceProp.major.deviceProp.minor] = [%d.%d]\n\n", deviceProp.major, deviceProp.minor);
    }

    // Loop to run the timing test multiple times:
    int kk;
    for (kk = 0; kk < NTESTS; kk++)
    {
        // We don't initialize randoms, because we want to compare different strategies:
        // Initializing random number generator:
        //  srand((unsigned)time(0));

        // Initializing the input array:
        for (int i = 0; i < NMAX; i++)
        {
            h_A[i] = (float)rand() / (float)RAND_MAX;
        }

        // Don't modify this: we need the double precision result to judge the accuracy:
        min0 = 0.0;
        for (int i = 0; i < NMAX; i++)
            min0 = min0 + (double)h_A[i];

        // Copying the data to device (we don't time it):
        if (error = hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, NMAX * sizeof(float), 0, hipMemcpyHostToDevice))
        {
            printf("Error %d\n", error);
            exit(error);
        }

        //--------------------------------------------------------------------------------
        if (error = hipDeviceSynchronize())
        {
            printf("Error %d\n", error);
            exit(error);
        }
        gettimeofday(&tdr0, NULL);

        // First level binary reduction:
        MyKernel1<<<NBLOCKS, BLOCK_SIZE>>>();

        // Second level binary reduction (only one block):
        MyKernel2<<<1, NBLOCKS>>>();

        // Copying the result back to host (we time it):
        if (error = hipMemcpyFromSymbol(&min, HIP_SYMBOL(d_min), sizeof(float), 0, hipMemcpyDeviceToHost))
        {
            printf("Error %d\n", error);
            exit(error);
        }

        if (error = hipDeviceSynchronize())
        {
            printf("Error %d\n", error);
            exit(error);
        }
        gettimeofday(&tdr1, NULL);
        tdr = tdr0;
        timeval_subtract(&restime, &tdr1, &tdr);

        printf("Min: %e (relative error %e)\n", min, fabs((double)min - min0) / min0);

        printf("Time: %e\n", restime);
        //--------------------------------------------------------------------------------

    } // kk loop

    return 0;
}
