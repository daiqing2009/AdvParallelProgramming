#include "hip/hip_runtime.h"
/* Atomic reduction solution.
 */

#include <sys/time.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

// Number of times to run the test (for better timings accuracy):
#define NTESTS 10

// Number of threads in one block (possible range is 32...1024):
#define BLOCK_SIZE 256

// Total number of threads (total number of elements to process in the kernel):
#define NMAX 512

#define NBLOCKS NMAX *(NMAX - 1) / BLOCK_SIZE / 2

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))
#define SQUARE(x) ((x) * (x))

// Input array (global host memory):
float h_X[NMAX];
float h_Y[NMAX];
float h_dist[NMAX * (NMAX - 1) / 2];

__device__ float d_X[NMAX];
__device__ float d_Y[NMAX];
__device__ float d_dist[NMAX * (NMAX - 1) / 2];
__device__ float d_min;

// It messes up with y!
int timeval_subtract(double *result, struct timeval *x, struct timeval *y);

//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
__device__ static float fatomicMin(float *addr, float value)
{
    float old = *addr, assumed;
    if (old <= value)
        return old;
    do
    {
        assumed = old;
        old = atomicCAS((unsigned int *)addr, __float_as_int(assumed), __float_as_int(value));
    } while (old != assumed);

    return old;
}

__global__ void init_kernel()
{
    d_min = 1.0;
    return;
}

// one tread per particle
__global__ void OneThreadPerParticleKernel()
{

    __shared__ float b_min[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float thread_min, next_dist = 1;

    // Not needed, because NMAX is a power of two:
    // if (i >= NMAX)
    //     return;
    
    // calculate the distance related to i and find the min of current thread
    for (int j = i + 1; j < BLOCK_SIZE; j += 2)
    {
        next_dist = sqrt(SQUARE(d_X[i] - d_X[j]) + SQUARE(d_Y[i] - d_Y[j]));
        thread_min = next_dist < thread_min ? next_dist : thread_min;
    }

    // To make sure all threads finished calc
    __syncthreads();

    // find the min within the block
    int nTotalThreads = blockDim.x; // Total number of active threads;
    // only the first half of the threads will be active.
    while (nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1); // divide by two
        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            b_min[threadIdx.x] = MIN(b_min[threadIdx.x], b_min[thread2]); // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint; // Reducing the binary tree size by two
    }

    // find the min among blocks, i.e. global min
    if (threadIdx.x == 0)
    {
        fatomicMin(&d_min, b_min[0]);
    }

    return;
}

__global__ void OneThreadPerPairKernel()
{
}

int main(int argc, char **argv)
{
    struct timeval tdr0, tdr1, tdr;
    double restime, min0;
    float min;
    int error;

    // hipMalloc((void **)&d_X, NMAX * sizeof(float));
    // hipMalloc((void **)&d_Y, NMAX * sizeof(float));
    // hipMalloc((void **)&d_dist, NMAX * (NMAX - 1) / 2 * sizeof(float));

    // Loop to run the timing test multiple times:
    for (int kk = 0; kk < NTESTS; kk++)
    {

        // We don't initialize randoms, because we want to compare different strategies:
        // Initializing random number generator:
        srand(kk);

        // Initializing the input array:
        for (int i = 0; i < NMAX; i++)
        {
            h_X[i] = (float)rand() / (float)RAND_MAX;
            h_Y[i] = (float)rand() / (float)RAND_MAX;
        }

        // Computer distances in a CPU serial function
        for (int i = NMAX - 1; i > 0; i--)
            for (int j = 0; j < NMAX; j++)
                h_dist[i * NMAX + j] = sqrt(SQUARE(h_X[i] - h_X[j]) + SQUARE(h_Y[i] - h_Y[j]));

        // Find the minimal in serial way
        min0 = 1;
        for (int i = 0; i < (NMAX - 1) * NMAX; i++)
            if (h_dist[i] < min0)
                min0 = (double)h_dist[i];

        // Copying the data to device (we don't time it):
        if (error = hipMemcpy(d_X, h_X, NMAX * sizeof(float), hipMemcpyHostToDevice))
        {
            printf("Error copy X to device %d\n", error);
            exit(error);
        }
        if (error = hipMemcpy(d_Y, h_Y, NMAX * sizeof(float), hipMemcpyHostToDevice))
        {
            printf("Error copy Y to device %d\n", error);
            exit(error);
        }

        init_kernel<<<1, 1>>>();
        if (error = hipDeviceSynchronize())
        {
            printf("Error %d\n", error);
            exit(error);
        }
        //--------------------------------------------------------------------------------
        gettimeofday(&tdr0, NULL);

        // Hybrid binary/atomic reduction:
        OneThreadPerParticleKernel<<<NBLOCKS, BLOCK_SIZE>>>();

        // thrust::device_ptr<float> d_ptr_A(d_A);
        // float reduction_sum = thrust::reduce(d_ptr_A, d_ptr_A + NMAX);

        gettimeofday(&tdr1, NULL);
        tdr = tdr0;
        timeval_subtract(&restime, &tdr1, &tdr);
        // Copying the result back to host (we time it):
        if (error = hipMemcpyFromSymbol(&min, HIP_SYMBOL(d_min), sizeof(float), 0, hipMemcpyDeviceToHost))
        {
            printf("Error %d\n", error);
            exit(error);
        }

        if (error = hipDeviceSynchronize())
        {
            printf("Error %d\n", error);
            exit(error);
        }
        printf("Min: %e (relative error %e)\n", min, fabs((double)min - min0) / min0);

        printf("Time: %e\n", restime);
        //--------------------------------------------------------------------------------

    } // kk loop

    // hipFree(d_X);
    // hipFree(d_Y);
    // hipFree(d_dist);

    return 0;
}

/* Subtract the `struct timeval' values X and Y,
   storing the result in RESULT.
   Return 1 if the difference is negative, otherwise 0.  */
int timeval_subtract(double *result, struct timeval *x, struct timeval *y)
{
    struct timeval result0;

    /* Perform the carry for the later subtraction by updating y. */
    if (x->tv_usec < y->tv_usec)
    {
        int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
        y->tv_usec -= 1000000 * nsec;
        y->tv_sec += nsec;
    }
    if (x->tv_usec - y->tv_usec > 1000000)
    {
        int nsec = (y->tv_usec - x->tv_usec) / 1000000;
        y->tv_usec += 1000000 * nsec;
        y->tv_sec -= nsec;
    }

    /* Compute the time remaining to wait.
       tv_usec is certainly positive. */
    result0.tv_sec = x->tv_sec - y->tv_sec;
    result0.tv_usec = x->tv_usec - y->tv_usec;
    *result = ((double)result0.tv_usec) / 1e6 + (double)result0.tv_sec;

    /* Return 1 if result is negative. */
    return x->tv_sec < y->tv_sec;
}